#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h> /* USHRT_MAX 상수를 위해서 사용한다. */
#include <math.h>
#include <iostream>
#include "bmpHeader.h"
/* 이미지 데이터의 경계 검사를 위한 매크로 */
#define LIMIT_UBYTE(n) ((n)>UCHAR_MAX)?UCHAR_MAX:((n)<0)?0:(n)
#define widthbytes(bits) (((bits)+31)/32*4)
typedef unsigned char ubyte;
//Cuda kernel for converting RGB image into a GreyScale image

__global__ void convertToBlur(ubyte *inimg, ubyte *out, int width, int height, int elemSize) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int z = threadIdx.z + blockIdx.z * blockDim.z;

	int size = width*elemSize;
	int offset = (x*elemSize+(y*size));

	unsigned char arr[25]={0,};
	float blur[5][5] = { {1/25.0, 1/25.0, 1/25.0, 1/25.0, 1/25.0},
		{1/25.0, 1/25.0, 1/25.0, 1/25.0, 1/25.0},
		{1/25.0, 1/25.0, 1/25.0, 1/25.0, 1/25.0},
		{1/25.0, 1/25.0, 1/25.0, 1/25.0, 1/25.0},
		{1/25.0, 1/25.0, 1/25.0, 1/25.0, 1/25.0} };

	// inSide
	float sum = 0.0;
	if ( (x > 1 && x < width-2) && (y >1 && y < height-2) ) {
		for(int i = -2; i < 3; i++) {
			for(int j = -2; j < 3; j++) {
				sum += blur[i+2][j+2]*inimg[(x+i)*elemSize+((y+j)*size)+z];
			}
		}
		out[offset +z] = LIMIT_UBYTE(sum);
	}

	//LeftVertex
	else if(x ==0){
		//LeftTopVertex
		if(y==0){
			arr[0] = arr[1]= arr[2] = arr[5] = arr[6] = arr[7] = arr[10] = arr[11] = arr[12] = inimg[offset+z];
			arr[3] = arr[8] = arr[13] = inimg[(x+1)*elemSize+(y*size)+z];
			arr[4] = arr[9] = arr[14] = inimg[(x+2)*elemSize+(y*size)+z];
			arr[15] = arr[16] = arr[17] = inimg[x*elemSize+((y+1)*size)+z];
			arr[20] = arr[21] = arr[22] = inimg[x*elemSize+((y+2)*size)+z];
			arr[18] = inimg[(x+1)*elemSize+((y+1)*size)+z];
			arr[19] = inimg[(x+2)*elemSize+((y+1)*size)+z];
			arr[23] = inimg[(x+1)*elemSize+((y+2)*size)+z];
			arr[24] = inimg[(x+2)*elemSize+((y+2)*size)+z];
		}
		else if(y == 1){
			arr[0] = arr[1] = arr[2] = arr[5] = arr[6] = arr[7] = inimg[x*elemSize+((y-1)*size)+z];
			arr[10] = arr[11] = arr[12] = inimg[offset+z];
			arr[3] = arr[8] = inimg[(x+1)*elemSize+((y-1)*size)+z];
			arr[4] = arr[9] = inimg[(x+2)*elemSize+((y-1)*size)+z];
			arr[13] = inimg[(x+1)*elemSize+(y*size)+z];
			arr[14] = inimg[(x+2)*elemSize+(y*size)+z];
			arr[15] = arr[16] = arr[17] = inimg[x*elemSize+((y+1)*size)+z];
			arr[18] = inimg[(x+1)*elemSize+((y+1)*size)+z];
			arr[19] = inimg[(x+2)*elemSize+((y+1)*size)+z];
			arr[20] = arr[21] = arr[22] = inimg[x*elemSize+((y+2)*size)+z];
			arr[23] = inimg[(x+1)*elemSize+((y+2)*size)+z];
			arr[24] = inimg[(x+2)*elemSize+((y+2)*size)+z];
		}
		else if(y==height-2){
			arr[0] = arr[1] = arr[2] = inimg[x*elemSize+((y-2)*size)+z];
			arr[3] = inimg[(x+1)*elemSize+((y-2)*size)+z];
			arr[4] = inimg[(x+2)*elemSize+((y-2)*size)+z];
			arr[5] = arr[6] = arr[7] = inimg[x*elemSize+((y-1)*size)+z];
			arr[8] = inimg[(x+1)*elemSize+((y-1)*size)+z];
			arr[9] = inimg[(x+2)*elemSize+((y-1)*size)+z];
			arr[10] = arr[11] = arr[12] = inimg[offset+z];
			arr[13] = inimg[(x+1)*elemSize+(y*size)+z];
			arr[14] = inimg[(x+2)*elemSize+(y*size)+z];
			arr[15] = arr[16] = arr[17] = arr[20] = arr[21] = arr[22] = inimg[x*elemSize+((y+1)*size)+z];
			arr[18] = arr[23] = inimg[(x+1)*elemSize+((y+1)*size)+z];
			arr[19] = arr[24] = inimg[(x+2)*elemSize+((y+1)*size)+z];
		}
		//LeftBottomVertex
		else if(y == height-1){
			arr[0] = arr[1] = arr[2] = inimg[x*elemSize+((y-2)*size)+z];
			arr[3] = inimg[(x+1)*elemSize+((y-2)*size)+z];
			arr[4] = inimg[(x+2)*elemSize+((y-2)*size)+z];
			arr[5] = arr[6] = arr[7] = inimg[x*elemSize+((y-1)*size)+z];
			arr[8] = inimg[(x+1)*elemSize+((y-1)*size)+z];
			arr[9] = inimg[(x+2)*elemSize+((y-1)*size)+z];
			arr[10] = arr[11]= arr[15] = arr[16] = arr[17] = arr[20] = arr[21] = arr[22] = arr[12] = inimg[offset+z];
			arr[13] = arr[18] = arr[23] = inimg[(x+1)*elemSize+((y+1)*size)+z];
			arr[14] = arr[19] = arr[24] = inimg[(x+2)*elemSize+((y+1)*size)+z];
		}
		//LeftSide
		else{
			arr[0] = arr[1] = arr[2] = inimg[x*elemSize+((y-2)*size)+z];
			arr[3] = inimg[(x+1)*elemSize+((y-2)*size)+z];
			arr[4] = inimg[(x+2)*elemSize+((y-2)*size)+z];
			arr[5] = arr[6] = arr[7] = inimg[x*elemSize+((y-1)*size)+z];
			arr[8] = inimg[(x+1)*elemSize+((y-1)*size)+z];
			arr[9] = inimg[(x+2)*elemSize+((y-1)*size)+z];
			arr[10] = arr[11] = arr[12] = inimg[offset+z];
			arr[13] = inimg[(x+1)*elemSize+(y*size)+z];
			arr[14] = inimg[(x+2)*elemSize+(y*size)+z];
			arr[15] = arr[16] = arr[17] = inimg[x*elemSize+((y+1)*size)+z];
			arr[18] = inimg[(x+1)*elemSize+((y+1)*size)+z];
			arr[19] = inimg[(x+2)*elemSize+((y+1)*size)+z];
			arr[20] = arr[21] = arr[22] = inimg[x*elemSize+((y+2)*size)+z];
			arr[23] = inimg[(x+1)*elemSize+((y+2)*size)+z];
			arr[24] = inimg[(x+2)*elemSize+((y+2)*size)+z];
		}

	}
	//LeftSide
	else if(x==1){
		//LeftTopVertex
		if(y==0){
			arr[0] = arr[1] = arr[5] = arr[6] = arr[10] = arr[11] = inimg[(x-1)*elemSize+(y*size)+z];
			arr[2] = arr[7] = arr[12] = inimg[offset+z];
			arr[3] = arr[8] = arr[13] = inimg[(x+1)*elemSize+(y*size)+z];
			arr[4] = arr[9] = arr[14] = inimg[(x+2)*elemSize+(y*size)+z];
			arr[15] = arr[16] = inimg[(x-1)*elemSize+((y+1)*size)+z];
			arr[17] = inimg[x*elemSize+((y+1)*size)+z];
			arr[18] = inimg[(x+1)*elemSize+((y+1)*size)+z];
			arr[19] = inimg[(x+2)*elemSize+((y+1)*size)+z];
			arr[20] = arr[21] = inimg[(x-1)*elemSize+((y+2)*size)+z];
			arr[22] = inimg[x*elemSize+((y+2)*size)+z];
			arr[23] = inimg[(x+1)*elemSize+((y+2)*size)+z];
			arr[24] = inimg[(x+2)*elemSize+((y+2)*size)+z];
		}
		else if(y==1){
			arr[0] = arr[1] = arr[5] = arr[6] = inimg[(x-1)*elemSize+((y-1)*size)+z];
			arr[2] = arr[7] = inimg[x*elemSize+((y-1)*size)+z];
			arr[3] = arr[8] = inimg[(x+1)*elemSize+((y-1)*size)+z];
			arr[4] = arr[9] = inimg[(x+2)*elemSize+((y-1)*size)+z];
			arr[10] = arr[11] = inimg[(x-1)*elemSize+(y*size)+z];
			arr[12] = inimg[offset+z];
			arr[13] = inimg[(x+1)*elemSize+(y*size)+z];
			arr[14] = inimg[(x+2)*elemSize+(y*size)+z];
			arr[15] = arr[16] = inimg[(x-1)*elemSize+((y+1)*size)+z];
			arr[17] = inimg[x*elemSize+((y+1)*size)+z];
			arr[18] = inimg[(x+1)*elemSize+((y+1)*size)+z];
			arr[19] = inimg[(x+2)*elemSize+((y+1)*size)+z];
			arr[20] = arr[21] = inimg[(x-1)*elemSize+((y+2)*size)+z];
			arr[22] = inimg[x*elemSize+((y+2)*size)+z];
			arr[23] = inimg[(x+1)*elemSize+((y+2)*size)+z];
			arr[24] = inimg[(x+1)*elemSize+((y+2)*size)+z];
		}
		//LeftBottomVertex
		else if(y == height -1){
			arr[0] = arr[1] = inimg[(x-1)*elemSize+((y-2)*size)+z];
			arr[2] = inimg[x*elemSize+((y-2)*size)+z];
			arr[3] = inimg[(x+1)*elemSize+((y-2)*size)+z];
			arr[4] = inimg[(x+2)*elemSize+((y-2)*size)+z];
			arr[5] = arr[6] = inimg[(x-1)*elemSize+((y-1)*size)+z];
			arr[7] = inimg[x*elemSize+((y-1)*size)+z];
			arr[8] = inimg[(x+1)*elemSize+((y-1)*size)+z];
			arr[9] = inimg[(x+2)*elemSize+((y-1)*size)+z];
			arr[10] = arr[11] = arr[15] = arr[16] = arr[20] = arr[21] = inimg[(x-1)*elemSize+(y*size)+z];
			arr[12] = arr[17] = arr[22] = inimg[offset+z];
			arr[13] = arr[18] = arr[23] = inimg[(x+1)*elemSize+(y*size)  +z];
			arr[14] = arr[19] = arr[24] = inimg[(x+2)*elemSize+(y*size)+z];
		}
		else if(y == height -2){
			arr[0] = arr[1] = inimg[(x-1)*elemSize+((y-2)*size)+z];
			arr[2] = inimg[x*elemSize+((y-2)*size)+z];
			arr[3] = inimg[(x+1)*elemSize+((y-2)*size)+z];
			arr[4] = inimg[(x+2)*elemSize+((y-2)*size)+z];
			arr[5] = arr[6] = inimg[(x-1)*elemSize+((y-1)*size)+z];
			arr[7] = inimg[x*elemSize+((y-1)*size)+z];
			arr[8] = inimg[(x+1)*elemSize+((y-1)*size)+z];
			arr[9] = inimg[(x+2)*elemSize+((y-1)*size)+z];
			arr[10] = arr[11] = inimg[(x-1)*elemSize+(y*size)+z];
			arr[12] = inimg[offset+z];
			arr[13] = inimg[(x+1)*elemSize+(y*size)+z];
			arr[14] = inimg[(x+2)*elemSize+(y*size)+z];
			arr[15] = arr[16] = arr[20] = arr[21] = inimg[(x-1)*elemSize+((y+1)*size)+z];
			arr[17] = arr[22] = inimg[x*elemSize+((y+1)*size)+z];
			arr[18] = arr[23] = inimg[(x+1)*elemSize+((y+1)*size)+z];
			arr[19] = arr[24] = inimg[(x+2)*elemSize+((y+1)*size)+z];
		}
		//LeftSide
		else{
			arr[0] = arr[1] = inimg[(x-1)*elemSize+((y-2)*size)+z];
			arr[2] = inimg[x*elemSize+((y-2)*size)+z];
			arr[3] = inimg[(x+1)*elemSize+((y-2)*size)+z];
			arr[4] = inimg[(x+2)*elemSize+((y-2)*size)+z];
			arr[5] = arr[6] = inimg[(x-1)*elemSize+((y-1)*size)+z];
			arr[7] = inimg[x*elemSize+((y-1)*size)+z];
			arr[8] = inimg[(x+1)*elemSize+((y-1)*size)+z];
			arr[9] = inimg[(x+2)*elemSize+((y-1)*size)+z];
			arr[10] = arr[11] = inimg[(x-1)*elemSize+(y*size)+z];
			arr[12] = inimg[offset+z];
			arr[13] = inimg[(x+1)*elemSize+(y*size)+z];
			arr[14] = inimg[(x+2)*elemSize+(y*size)+z];
			arr[15] = arr[16] = inimg[(x-1)*elemSize+((y+1)*size)+z];
			arr[17] = inimg[x*elemSize+((y+1)*size)+z];
			arr[18] = inimg[(x+1)*elemSize+((y+1)*size)+z];
			arr[19] = inimg[(x+2)*elemSize+((y+1)*size)+z];
			arr[20] = arr[21] = inimg[(x-1)*elemSize+((y+2)*size)+z];
			arr[22] = inimg[x*elemSize+((y+2)*size)+z];
			arr[23] = inimg[(x+1)*elemSize+((y+2)*size)+z];
			arr[24] = inimg[(x+2)*elemSize+((y+2)*size)+z];
		}

	}
	//RightSide
	else if(x==width-2){
		//RightTopVertex
		if(y==0){
			arr[0] = arr[5] = arr[10] = inimg[(x-2)*elemSize+(y*size)+z];
			arr[1] = arr[6] = arr[11] = inimg[(x-1)*elemSize+(y*size)+z];
			arr[2] = arr[7] = arr[12] = inimg[offset+z];
			arr[3] = arr[8] = arr[4] = arr[9] = arr[14] = arr[13] = inimg[(x+1)*elemSize+(y*size)+z];
			arr[15] = inimg[(x-2)*elemSize+((y+1)*size)+z];
			arr[16] = inimg[(x-1)*elemSize+((y+1)*size)+z];
			arr[17] = inimg[x*elemSize+((y+1)*size)+z];
			arr[18] = arr[19] = inimg[(x+1)*elemSize+((y+1)*size)+z];
			arr[20] = inimg[(x-2)*elemSize+((y+2)*size)+z];
			arr[21] = inimg[(x-1)*elemSize+((y+2)*size)+z];
			arr[22] = inimg[x*elemSize+((y+2)*size)+z];
			arr[23] = arr[24] = inimg[(x+1)*elemSize+((y+2)*size)+z];
		}
		else if(y==1){
			arr[0] = arr[5] = inimg[(x-2)*elemSize+((y-1)*size)+z];
			arr[1] = arr[6] = inimg[(x-1)*elemSize+((y-1)*size)+z];
			arr[2] = arr[7] = inimg[x*elemSize+((y-1)*size)+z];
			arr[3] = arr[4] = arr[8] = arr[9] = inimg[(x+1)*elemSize+((y-1)*size)+z];
			arr[10] = inimg[(x-2)*elemSize+(y*size)+z];
			arr[11] = inimg[(x-1)*elemSize+(y*size)+z];
			arr[12] = inimg[offset+z];
			arr[13] = arr[14] = inimg[(x+1)*elemSize+(y*size)+z];
			arr[15] = inimg[(x-2)*elemSize+((y+1)*size)+z];
			arr[16] = inimg[(x-1)*elemSize+((y+1)*size)+z];
			arr[17] = inimg[x*elemSize+((y+1)*size)+z];
			arr[18] = arr[19] = inimg[(x+1)*elemSize+((y+1)*size)+z];
			arr[20] = inimg[(x-2)*elemSize+((y+2)*size)+z];
			arr[21] = inimg[(x-1)*elemSize+((y+2)*size)+z];
			arr[22] = inimg[x*elemSize+((y+2)*size)+z];
			arr[23] = arr[24] = inimg[(x+1)*elemSize+((y+2)*size)+z];
		}
		//RightBottomVertex
		else if(y==height-2){
			arr[0] = inimg[(x-2)*elemSize+((y-2)*size)+z];
			arr[1] = inimg[(x-1)*elemSize+((y-2)*size)+z];
			arr[2] = inimg[x*elemSize+((y-2)*size)+z];
			arr[3] = arr[4] = inimg[(x+1)*elemSize+((y-2)*size)+z];
			arr[5] = inimg[(x-2)*elemSize+((y-1)*size)+z];
			arr[6] = inimg[(x-1)*elemSize+((y-1)*size)+z];
			arr[7] = inimg[x*elemSize+((y-1)*size)+z];
			arr[8] = arr[9] = inimg[(x+1)*elemSize+((y-1)*size)+z];
			arr[10] = inimg[(x-2)*elemSize+(y*size)+z];
			arr[11] = inimg[(x-1)*elemSize+(y*size)+z];
			arr[12] = inimg[offset+z];
			arr[13] = arr[14] = inimg[(x+1)*elemSize+(y*size)+z];
			arr[15] = arr[20] = inimg[(x-2)*elemSize+((y+1)*size)+z];
			arr[16] = arr[21] = inimg[(x-1)*elemSize+((y+1)*size)+z];
			arr[17] = arr[22] = inimg[x*elemSize+((y+1)*size)+z];
			arr[18] = arr[23] = inimg[(x+1)*elemSize+((y+1)*size)+z];
			arr[19] = arr[24] = inimg[(x+2)*elemSize+((y+1)*size)+z];
		}
		else if(y == height-1){
			arr[0] = inimg[(x-2)*elemSize+((y-2)*size)+z];
			arr[1] = inimg[(x-1)*elemSize+((y-2)*size)+z];
			arr[2] = inimg[x*elemSize+((y-2)*size)+z];
			arr[3] = arr[4] = inimg[(x+1)*elemSize+((y-2)*size)+z];
			arr[5] = inimg[(x-2)*elemSize+((y-1)*size)+z];
			arr[6] = inimg[(x-1)*elemSize+((y-1)*size)+z];
			arr[7] = inimg[x*elemSize+((y-1)*size)+z];
			arr[8] = arr[9] = inimg[(x+1)*elemSize+((y-1)*size)+z];
			arr[10] = arr[15] = arr[20] = inimg[(x-2)*elemSize+(y*size)+z];
			arr[11] = arr[16] = arr[21] = inimg[(x-1)*elemSize+(y*size)+z];
			arr[12] = arr[17] = arr[22] = inimg[offset+z];
			arr[13] = arr[14] = arr[18] = arr[19] = arr[23] = arr[24] = inimg[(x+1)*elemSize+(y*size)+z];
		}
		//RightSide
		else{
			arr[0] = inimg[(x-2)*elemSize+((y-2)*size)+z];
			arr[1] = inimg[(x-1)*elemSize+((y-2)*size)+z];
			arr[2] = inimg[x*elemSize+((y-2)*size)+z];
			arr[3] = arr[4] = inimg[(x+1)*elemSize+((y-2)*size)+z];
			arr[5] = inimg[(x-2)*elemSize+((y-1)*size)+z];
			arr[6] = inimg[(x-1)*elemSize+((y-1)*size)+z];
			arr[7] = inimg[x*elemSize+((y-1)*size)+z];
			arr[8] = arr[9] = inimg[(x+1)*elemSize+((y-1)*size)+z];
			arr[10] = inimg[(x-2)*elemSize+(y*size)+z];
			arr[11] = inimg[(x-1)*elemSize+(y*size)+z];
			arr[12] = inimg[offset+z];
			arr[13] = arr[14] = inimg[(x+2)*elemSize+(y*size)+z];
			arr[15] = inimg[(x-2)*elemSize+((y+1)*size)+z];
			arr[16] = inimg[(x-1)*elemSize+((y+1)*size)+z];
			arr[17] = inimg[x*elemSize+((y+1)*size)+z];
			arr[18] = arr[19] = inimg[(x+1)*elemSize+((y+1)*size)+z];
			arr[20] = inimg[(x-2)*elemSize+((y+2)*size)+z];
			arr[21] = inimg[(x-1)*elemSize+((y+2)*size)+z];
			arr[22] = inimg[x*elemSize+((y+2)*size)+z];
			arr[23] = arr[24] = inimg[(x+1)*elemSize+((y+2)*size)+z];
		}
	}
	//RightSide
	else if(x==width-1){
		//RightTopVertex
		if(y==0){
			arr[0] = arr[5] = arr[10] = inimg[(x-2)*elemSize+(y*size)+z];
			arr[1] = arr[6] = arr[11] = inimg[(x-1)*elemSize+(y*size)+z];
			arr[2] = arr[7] = arr[3] = arr[8] = arr[13] = arr[4] = arr[9] = arr[14] = arr[12] = inimg[offset+z];
			arr[15] = inimg[(x-2)*elemSize+((y+1)*size)+z];
			arr[16] = inimg[(x-1)*elemSize+((y+1)*size)+z];
			arr[17] = arr[18] = arr[19] = inimg[x*elemSize+((y+1)*size)+z];
			arr[20] = inimg[(x-2)*elemSize+((y+2)*size)+z];
			arr[21] = inimg[(x-1)*elemSize+((y+2)*size)+z];
			arr[22] = arr[23] = arr[24] = inimg[x*elemSize+((y+2)*size)+z];
		}
		else if(y==1){
			arr[0] = arr[5] = inimg[(x-2)*elemSize+((y-1)*size)+z];
			arr[1] = arr[6] = inimg[(x-1)*elemSize+((y-1)*size)+z];
			arr[2] = arr[7] = arr[3] = arr[4] = arr[8] = arr[9] = inimg[x*elemSize+((y-1)*size)+z];
			arr[10] = inimg[(x-2)*elemSize+(y*size)+z];
			arr[11] = inimg[(x-1)*elemSize+(y*size)+z];
			arr[12] = arr[13] = arr[14] = inimg[offset+z];
			arr[15] = inimg[(x+1)*elemSize+((y+1)*size)+z];
			arr[16] = inimg[(x-1)*elemSize+((y+1)*size)+z];
			arr[17] = arr[18] = arr[19] = inimg[x*elemSize+((y+1)*size)+z];
			arr[20] = inimg[(x-2)*elemSize+((y+2)*size)+z];
			arr[21] = inimg[(x-1)*elemSize+((y+2)*size)+z];
			arr[22] = arr[23] = arr[24] = inimg[x*elemSize+((y+2)*size)+z];
		}
		//RightBottomVertex
		else if(y==height-1){
			arr[0] = inimg[(x-2)*elemSize+((y-2)*size)+z];
			arr[1] = inimg[(x-1)*elemSize+((y-2)*size)+z];
			arr[2] = arr[3] = arr[4] = inimg[x*elemSize+((y-2)*size)+z];
			arr[5] = inimg[(x-2)*elemSize+((y-1)*size)+z];
			arr[6] = inimg[(x-1)*elemSize+((y-1)*size)+z];
			arr[7] = arr[8] = arr[9] = inimg[x*elemSize+((y-1)*size)+z];
			arr[10] = arr[15] = arr[20] = inimg[(x-2)*elemSize+(y*size)+z];
			arr[11] = arr[16] = arr[21] = inimg[(x-1)*elemSize+(y*size)+z];
			arr[12] = arr[13] = arr[14] = arr[17] = arr[18] = arr[19] = arr[22] = arr[23] = arr[24] = inimg[offset+z];
		}
		else if(y==height-2){
			arr[0] = inimg[(x-2)*elemSize+((y-2)*size)+z];
			arr[1] = inimg[(x-1)*elemSize+((y-2)*size)+z];
			arr[2] = arr[3] = arr[4] = inimg[x*elemSize+((y-2)*size)+z];
			arr[5] = inimg[(x-2)*elemSize+((y-1)*size)+z];
			arr[6] = inimg[(x-1)*elemSize+((y-1)*size)+z];
			arr[7] = arr[8] = arr[9] = inimg[x*elemSize+((y-1)*size)+z];
			arr[10] = inimg[(x-2)*elemSize+(y*size)+z];
			arr[11] = inimg[(x-1)*elemSize+(y*size)+z];
			arr[12] = arr[13] = arr[14] = inimg[offset+z];
			arr[15] = arr[20] = inimg[(x-2)*elemSize+((y+1)*size)+z];
			arr[16] = arr[21] = inimg[(x-1)*elemSize+((y+1)*size)+z];
			arr[17] = arr[18] = arr[19] = arr[22] = arr[23] = arr[24] = inimg[x*elemSize+((y+1)*size)+z];
		}
		//RightSide
		else{
			arr[0] = inimg[(x-2)*elemSize+((y-2)*size)+z];
			arr[1] = inimg[(x-1)*elemSize+((y-2)*size)+z];
			arr[2] = arr[3] = arr[4] = inimg[x*elemSize+((y-2)*size)+z];
			arr[5] = inimg[(x-2)*elemSize+((y-1)*size)+z];
			arr[6] = inimg[(x-1)*elemSize+((y-1)*size)+z];
			arr[7] = arr[8] = arr[9] = inimg[x*elemSize+((y-1)*size)+z];
			arr[10] = inimg[(x-2)*elemSize+(y*size)+z];
			arr[11] = inimg[(x-1)*elemSize+(y*size)+z];
			arr[12] = arr[13] = arr[14] = inimg[offset+z];
			arr[15] = inimg[(x-2)*elemSize+((y+1)*size)+z];
			arr[16] = inimg[(x-1)*elemSize+((y+1)*size)+z];
			arr[17] = arr[18] = arr[19] = inimg[x*elemSize+((y+1)*size)+z];
			arr[20] = inimg[(x-2)*elemSize+((y+2)*size)+z];
			arr[21] = inimg[(x-1)*elemSize+((y+2)*size)+z];
			arr[22] = arr[23] = arr[24] = inimg[x*elemSize+((y+2)*size)+z];

		}
	}

	//TopSide
	else if( y==0){
		if(x>1 && x <width-2){
			arr[0] = arr[5] = arr[10] = inimg[(x-2)*elemSize+(y*size)+z];
			arr[1] = arr[6] = arr[11] = inimg[(x-1)*elemSize+(y*size)+z];
			arr[2] = arr[7] = arr[12] = inimg[offset+z];
			arr[3] = arr[8] = arr[13] = inimg[(x+1)*elemSize+(y*size)+z];
			arr[4] = arr[9] = arr[14] = inimg[(x+2)*elemSize+(y*size)+z];
			arr[15] = inimg[(x-2)*elemSize+((y+1)*size)+z];
			arr[16] = inimg[(x-1)*elemSize+((y+1)*size)+z];
			arr[17] = inimg[x*elemSize+((y+1)*size)+z];
			arr[18] = inimg[(x+1)*elemSize+((y+1)*size)+z];
			arr[19] = inimg[(x+2)*elemSize+((y+1)*size)+z];
			arr[20] = inimg[(x-2)*elemSize+((y+2)*size)+z];
			arr[21] = inimg[(x-1)*elemSize+((y+2)*size)+z];
			arr[22] = inimg[x*elemSize+((y+2)*size)+z];
			arr[23] = inimg[(x+1)*elemSize+((y+2)*size)+z];
			arr[24] = inimg[(x+2)*elemSize+((y+2)*size)+z];
		}
	}
	else if(y==1){
		if(x>1 && x <width-2){
			arr[0] = arr[5] = inimg[(x-2)*elemSize+((y-1)*size)+z];
			arr[1] = arr[6] = inimg[(x-1)*elemSize+((y-1)*size)+z];
			arr[2] = arr[7] = inimg[x*elemSize+((y-1)*size)+z];
			arr[3] = arr[8] = inimg[(x+1)*elemSize+((y-1)*size)+z];
			arr[4] = arr[9] = inimg[(x+2)*elemSize+((y-1)*size)+z];
			arr[10] = inimg[(x-2)*elemSize+(y*size)+z];
			arr[11] = inimg[(x-1)*elemSize+(y*size)+z];
			arr[12] = inimg[offset+z];
			arr[13] = inimg[(x+1)*elemSize+(y*size)+z];
			arr[14] = inimg[(x+2)*elemSize+(y*size)+z];
			arr[15] = inimg[(x-2)*elemSize+((y+1)*size)+z];
			arr[16] = inimg[(x-1)*elemSize+((y+1)*size)+z];
			arr[17] = inimg[x*elemSize+((y+1)*size)+z];
			arr[18] = inimg[(x+1)*elemSize+((y+1)*size)+z];
			arr[19] = inimg[(x+2)*elemSize+((y+1)*size)+z];
			arr[20] = inimg[(x-2)*elemSize+((y+2)*size)+z];
			arr[21] = inimg[(x-1)*elemSize+((y+2)*size)+z];
			arr[22] = inimg[x*elemSize+((y+2)*size)+z];
			arr[23] = inimg[(x+1)*elemSize+((y+2)*size)+z];
			arr[24] = inimg[(x+2)*elemSize+((y+2)*size)+z];
		}
	}
	//BottomSide
	else if(y==height-2){
		if(x>1 && x <width-2){
			arr[0] = inimg[(x-2)*elemSize+((y-2)*size)+z];
			arr[1] = inimg[(x-1)*elemSize+((y-2)*size)+z];
			arr[2] = inimg[x*elemSize+((y-2)*size)+z];
			arr[3] = inimg[(x+1)*elemSize+((y-2)*size)+z];
			arr[4] = inimg[(x+2)*elemSize+((y-2)*size)+z];
			arr[5] = inimg[(x-2)*elemSize+((y-1)*size)+z];
			arr[6] = inimg[(x-1)*elemSize+((y-1)*size)+z];
			arr[7] = inimg[x*elemSize+((y-1)*size)+z];
			arr[8] = inimg[(x+1)*elemSize+((y-1)*size)+z];
			arr[9] = inimg[(x+2)*elemSize+((y-1)*size)+z];
			arr[10] = inimg[(x-2)*elemSize+(y*size)+z];
			arr[11] = inimg[(x-1)*elemSize+(y*size)+z];
			arr[12] = inimg[offset+z];
			arr[13] = inimg[(x+1)*elemSize+(y*size)+z];
			arr[14] = inimg[(x+2)*elemSize+(y*size)+z];
			arr[15] = arr[20] = inimg[(x-2)*elemSize+((y+1)*size)+z];
			arr[16] = arr[21] = inimg[(x-1)*elemSize+((y+1)*size)+z];
			arr[17] = arr[22] = inimg[x*elemSize+((y+1)*size)+z];
			arr[18] = arr[23] = inimg[(x+1)*elemSize+((y+1)*size)+z];
			arr[19] = arr[24] = inimg[(x+2)*elemSize+((y+1)*size)+z];
		}	
	}
	else if( y==height-1){
		if(x>1 && x <width-2){
			arr[0] = inimg[(x-2)*elemSize+((y-2)*size)+z];
			arr[1] = inimg[(x-1)*elemSize+((y-2)*size)+z];
			arr[2] = inimg[x*elemSize+((y-2)*size)+z];
			arr[3] = inimg[(x+1)*elemSize+((y-2)*size)+z];
			arr[4] = inimg[(x+2)*elemSize+((y-2)*size)+z];
			arr[5] = inimg[(x-2)*elemSize+((y-1)*size)+z];
			arr[6] = inimg[(x-1)*elemSize+((y-1)*size)+z];
			arr[7] = inimg[x*elemSize+((y-1)*size)+z];
			arr[8] = inimg[(x+1)*elemSize+((y-1)*size)+z];
			arr[9] = inimg[(x+2)*elemSize+((y-1)*size)+z];
			arr[10] = arr[15] = arr[20] = inimg[(x-2)*elemSize+(y*size)+z];
			arr[11] = arr[16] = arr[21] = inimg[(x-1)*elemSize+(y*size)+z];
			arr[12] = arr[17] = arr[22] = inimg[offset+z];
			arr[13] = arr[18] = arr[23] = inimg[(x+1)*elemSize+(y*size)+z];
			arr[14] = arr[19] = arr[24] = inimg[(x+2)*elemSize+(y*size)+z];
		}
	}

	int cnt=0;
	for(int i = -2; i < 3; i++) {
		for(int j = -2; j < 3; j++) {
			sum += blur[i+2][j+2]*arr[cnt++];
		}
	}
	out[offset+z] = LIMIT_UBYTE(sum);
}

int main(int argc, char** argv)
{
	FILE* fp;
	BITMAPFILEHEADER bmpHeader; /* BMP FILE INFO */
	BITMAPINFOHEADER bmpInfoHeader; /* BMP IMAGE INFO */
	//RGBQUAD *palrgb;
	ubyte *inimg, *outimg;
	if(argc != 3) {
		fprintf(stderr, "usage : %s input.bmp output.bmp\n", argv[0]);
		return -1;
	}
	/***** read bmp *****/
	if((fp=fopen(argv[1], "rb")) == NULL) {
		fprintf(stderr, "Error : Failed to open file...₩n");
		return -1;
	}
	/* BITMAPFILEHEADER 구조체의 데이터 */
	fread(&bmpHeader, sizeof(BITMAPFILEHEADER), 1, fp);
	/* BITMAPINFOHEADER 구조체의 데이터 */
	fread(&bmpInfoHeader, sizeof(BITMAPINFOHEADER), 1, fp);
	/* 트루 컬러를 지원하면 변환할 수 없다. */
	if(bmpInfoHeader.biBitCount != 24) {
		perror("This image file doesn't supports 24bit color\n");
		fclose(fp);
		return -1;
	}

	int elemSize = bmpInfoHeader.biBitCount/8.;
	int stride = bmpInfoHeader.biWidth * elemSize;
	//widthbytes(bits) (((bits)+31)/32*4)
	int imageSize = stride * bmpInfoHeader.biHeight;

	inimg = (ubyte*)malloc(sizeof(ubyte)*imageSize);
	outimg = (ubyte*)malloc(sizeof(ubyte)*imageSize);

	fread(inimg, sizeof(ubyte), imageSize, fp);
	fclose(fp);

	ubyte *d_inimg = NULL, *d_outimg = NULL;
	//allocate and initialize memory on device
	hipMalloc(&d_inimg, sizeof(ubyte) * imageSize);
	hipMalloc(&d_outimg, sizeof(ubyte) * imageSize);
	hipMemset(d_outimg, 0, sizeof(ubyte) * imageSize);
	//copy host rgb data array to device rgb data array
	hipMemcpy(d_inimg, inimg, sizeof(ubyte) * imageSize, hipMemcpyHostToDevice);

	//define block and grid dimensions
	const dim3 dimGrid((int)ceil((bmpInfoHeader.biWidth/32)), (int)ceil((bmpInfoHeader.biHeight)/4),1);
	const dim3 dimBlock(32, 4, elemSize);

	//execute cuda kernel
	convertToBlur<<<dimGrid, dimBlock>>>(d_inimg, d_outimg, bmpInfoHeader.biHeight, bmpInfoHeader.biWidth, elemSize);
	//copy computed gray data array from device to host
	hipMemcpy(outimg, d_outimg, sizeof(ubyte) * imageSize, hipMemcpyDeviceToHost);

	hipFree(d_outimg);
	hipFree(d_inimg);

	/***** write bmp *****/
	if((fp=fopen(argv[2], "wb"))==NULL) {
		fprintf(stderr, "Error : Failed to open file...₩n");
		return -1;
	}
	/*
	   palrgb = (RGBQUAD*)malloc(sizeof(RGBQUAD)*256);
	   for(int x = 0; x < 256; x++) {
	   palrgb[x].rgbBlue = palrgb[x].rgbGreen = palrgb[x].rgbRed = x;
	   palrgb[x].rgbReserved = 0;
	   }
	 */
	bmpInfoHeader.biBitCount = 24;
	bmpInfoHeader.SizeImage = imageSize;
	//bmpInfoHeader.biCompression = 0;
	//bmpInfoHeader.biClrUsed = 0;
	//bmpInfoHeader.biClrImportant = 0;
	//bmpHeader.bfOffBits = sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER) + sizeof(RGBQUAD)*256;
	bmpHeader.bfSize = bmpInfoHeader.SizeImage;
	/* BITMAPFILEHEADER 구조체의 데이터 */
	fwrite(&bmpHeader, sizeof(BITMAPFILEHEADER), 1, fp);
	/* BITMAPINFOHEADER 구조체의 데이터 */
	fwrite(&bmpInfoHeader, sizeof(BITMAPINFOHEADER), 1, fp);
	//fwrite(palrgb, sizeof(RGBQUAD), 256, fp);
	//fwrite(inimg, sizeof(ubyte), imageSize, fp);
	fwrite(outimg, sizeof(ubyte), imageSize, fp);
	fclose(fp);
	free(inimg);
	free(outimg);

	printf("Success blur\n");
	return 0;
}
